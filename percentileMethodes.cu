#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <random>

#define CHECK_CUDA(expr)                                                     \
  do {                                                                       \
    hipError_t _err = (expr);                                               \
    if (_err != hipSuccess) {                                               \
      std::cerr << "CUDA error " << hipGetErrorString(_err)                 \
                << " at " << __FILE__ << ":" << __LINE__ << std::endl;       \
      std::exit(EXIT_FAILURE);                                               \
    }                                                                        \
  } while (0)


/**
 * @brief Computes the linear interpolation between 2 input values
 *
 * @param a the first value
 * @param b the second value
 * @param t interpolation parameter (between 0 and 1)
 *
 * @return the interpolated value
 */
inline float lerp_two(float a, float b, double t)
{
	return fmaf(static_cast<float>(t), (b - a), a);
}

/**
 * @brief Sort the input buffer dopplerVolume and computes the percentile with Thrust
 *        see: https://nvidia.github.io/cccl/thrust/api/function_group__sorting_1gad39e37d88f8334cbdd3e047a53e5cfba.html
 *
 * @param dopplerVolume the input buffer (memory on the host)
 * @param percentil the percentile rank
 *
 * @return the percentile score
 */
float percentileWithThrust(const std::vector<float>& dopplerVolume,
                           float percentil)
{
  unsigned int volumeSize = dopplerVolume.size();

	thrust::device_vector<float> d_dopplerVolume(dopplerVolume);

	const double r = percentil * (volumeSize - 1);
	const size_t k = static_cast<size_t>(floor(r));
	const double frac = r - static_cast<double>(k);

	thrust::sort(thrust::cuda::par.on(0), d_dopplerVolume.begin(), d_dopplerVolume.end()); // execute on stream 0
	float vk = d_dopplerVolume[k];
	float vk1 = (k + 1 < volumeSize) ? d_dopplerVolume[k + 1] : vk;

	const float res = (frac == 0.0 || k + 1 == volumeSize) ? vk : lerp_two(vk, vk1, frac);

  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipDeviceSynchronize());

  return res;
}

/**
 * @brief Sort the input buffer dopplerVolume and computes the percentile with Radix
 *        see: https://nvidia.github.io/cccl/cub/api/structcub_1_1DeviceRadixSort.html
 *
 * @param dopplerVolume the input buffer (memory on the host)
 * @param percentil the percentile rank
 *
 * @return the percentile score
 */
float percentileWithDeviceRadixSort(const std::vector<float>& dopplerVolume, float percentil)
{
  unsigned int volumeSize = dopplerVolume.size();
  float *d_in = nullptr;
  float *d_out = nullptr;

  CHECK_CUDA(hipMalloc(&d_in, volumeSize * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_out, volumeSize * sizeof(float)));

  // Copy input data
  CHECK_CUDA(hipMemcpy(d_in, dopplerVolume.data(),
                        volumeSize * sizeof(float),
                        hipMemcpyHostToDevice));

  // Temporary storage
  void *d_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;

  // Request temp storage size
  hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_in, d_out, volumeSize);

  // Allocate temp storage
  CHECK_CUDA(hipMalloc(&d_temp_storage, temp_storage_bytes));

  // Run sorting
  hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_in, d_out, volumeSize);

	const double r = percentil * (volumeSize - 1);
	const size_t k = static_cast<size_t>(floor(r));
	const double frac = r - static_cast<double>(k);

  // Retrieve the k-nth and the k-nth+1 values from the device.
  // It is more efficient to retrieve just those 2 values from the sorted buffer rather than copying the
  // full sorted buffer on the host to retrieve those values.
  // I guess this mimics the behavior of the thrust::device_vector accessors.
  float valueK;
  float valueK1;

  CHECK_CUDA(hipMemcpy(&valueK, d_out + k, sizeof(float), hipMemcpyDeviceToHost));
  CHECK_CUDA(hipMemcpy(&valueK1, d_out + k + 1, sizeof(float), hipMemcpyDeviceToHost));

	float vk = valueK;
	float vk1 = (k + 1 < volumeSize) ? valueK1 : vk;

	const float res = (frac == 0.0 || k + 1 == volumeSize) ? vk : lerp_two(vk, vk1, frac);

  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipDeviceSynchronize());

  // Cleanup
  hipFree(d_in);
  hipFree(d_out);
  hipFree(d_temp_storage);

  return res;
}

/**
 * @brief Program to test the timing of the computation of the percentile using the sorting method of
 *        both Thrust and Radix for comparison.
 */
int main() {

    size_t freeB, totalB;
    CHECK_CUDA(hipMemGetInfo(&freeB, &totalB));
    std::fprintf(stderr, "GPU free %.2f GiB / %.2f GiB\n",
             freeB / (1024.0*1024*1024), totalB / (1024.0*1024*1024));

    // Generate noise on the host
    // const unsigned int volumeSize = 80 * 80 * 146 * 300; // For real size test
    const unsigned int volumeSize = 80; // For quick test
    std::vector<float> h_noise(volumeSize);

    std::mt19937 rng(12345); // deterministic seed
    std::uniform_real_distribution<float> dist(0.0f, 1.0f);

    for (unsigned int i = 0; i < volumeSize; ++i) {
        // Simple white noise in [0,1); replace with any noise you like
        h_noise[i] = dist(rng);
        //std::cout << "i :" << i << " value: " << h_noise[i] << std::endl;
    }

    float percentileRank {0.9f}; // Must be between 0 and 1

    std::cout << "Method Thrust: " << percentileWithThrust(h_noise, percentileRank) << std::endl;
    std::cout << "Method Radix:  " << percentileWithDeviceRadixSort(h_noise, percentileRank) << std::endl;

    return 0;
}